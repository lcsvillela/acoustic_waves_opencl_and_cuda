#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>


typedef double my_type;

void print_time(const char* label) {
    struct timeval tv;
    gettimeofday(&tv, NULL);

    struct tm *gmt = gmtime(&tv.tv_sec);

    printf("%s: %04d-%02d-%02d %02d:%02d:%02d.%06ld UTC\n", label,
           gmt->tm_year + 1900, gmt->tm_mon + 1, gmt->tm_mday,
           gmt->tm_hour, gmt->tm_min, gmt->tm_sec, tv.tv_usec);
}


__global__ void initialize_u(my_type *u, int N, int M)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < N && y < M)
    {
        u[x + y * N] = 0.0;
    }
}

__global__ void define_c(my_type *c, my_type c_max, int N, int M)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < N && y < M)
    {
        c[x + y * N] = c_max;
    }
}

__global__ void define_initial_condition(my_type *u, int N, int M, int x0, int y0)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < N && y < M)
    {
        my_type a = 0.2;
        u[x + y * N] = exp(-a * ((x - x0) * (x - x0) + (y - y0) * (y - y0)));
    }
}

__global__ void calculate_wave(my_type *u_next, my_type *u_cur, my_type *u_prev, my_type *c, int N, int M, my_type dt, my_type dh)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x > 0 && x < N - 1 && y > 0 && y < M - 1)
    {
        my_type alpha = c[x + y * N] * c[x + y * N] * dt * dt / (dh * dh);
        u_next[x + y * N] = 2 * u_cur[x + y * N] - u_prev[x + y * N] +
                            alpha * (u_cur[(x + 1) + y * N] - 2 * u_cur[x + y * N] + u_cur[(x - 1) + y * N] +
                                     u_cur[x + (y + 1) * N] - 2 * u_cur[x + y * N] + u_cur[x + (y - 1) * N]);
    }
}

void ricker(my_type f0, my_type t0, my_type* time_arr, my_type* s_arr, int N_time)
{
    my_type arg;
    for (int i = 0; i < N_time; i++)
    {
        arg = M_PI * f0 * (time_arr[i] - t0);
        s_arr[i] = (2 * arg * arg - 1) * exp(-arg * arg);
    }
}



int main(int argc, char *argv[])
{
    
    int threads = atoi(argv[1]);
    int DIMENSION = atoi(argv[2]);
    sleep(2);
    const int N = DIMENSION; // Número de elementos espaciais na direção x
    const int M = DIMENSION; // Número de elementos espaciais na direção y
    const my_type dh = 1; // Passo espacial [m]
    const my_type c_max = 3000.0; // Velocidade máxima [m/s]
    const my_type dt = (dh / c_max) * 0.1; // Passo temporal [s]
    const my_type time_duration = 8; // Duração total da simulação [s]
    const int N_time = (int)(time_duration / dt); // Número de elementos temporais
    const int x0 = N / 2; // Posição da fonte na direção x
    const int y0 = M / 2; // Posição da fonte na direção y

    my_type *time_arr = (my_type*) malloc(sizeof(my_type) * N_time);
    my_type *s_arr = (my_type*) malloc(sizeof(my_type) * N_time);
    my_type *h_u = (my_type*) malloc(sizeof(my_type) * N * M);

    for (int i = 0; i < N_time; i++)
    {
        time_arr[i] = i * dt;
    }
    my_type f0 = 100.0; // Frequência da onda de Ricker [Hz]
    my_type t0 = 0.01; // Deslocamento da onda de Ricker [s]
    ricker(f0, t0, time_arr, s_arr, N_time);

    my_type *d_u_next, *d_u_cur, *d_u_prev, *d_c;
    hipMalloc((void**)&d_u_next, sizeof(my_type) * N * M);
    hipMalloc((void**)&d_u_cur, sizeof(my_type) * N * M);
    hipMalloc((void**)&d_u_prev, sizeof(my_type) * N * M);
    hipMalloc((void**)&d_c, sizeof(my_type) * N * M);

    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t initialize_u_start1, initialize_u_stop1, define_c_start, define_c_stop, define_initial_condition_start, define_initial_condition_stop, calculate_wave_start, calculate_wave_stop, initialize_u_start2, initialize_u_stop2, initialize_u_start3, initialize_u_stop3;
    hipEventCreate(&initialize_u_start1);
    hipEventCreate(&initialize_u_stop1);
    hipEventCreate(&initialize_u_start2);
    hipEventCreate(&initialize_u_stop2);
    hipEventCreate(&initialize_u_start3);
    hipEventCreate(&initialize_u_stop3);
    hipEventCreate(&define_c_start);
    hipEventCreate(&define_c_stop);
    hipEventCreate(&define_initial_condition_start);
    hipEventCreate(&define_initial_condition_stop);
    hipEventCreate(&calculate_wave_start);
    hipEventCreate(&calculate_wave_stop);

    print_time("inicio: ");
    hipEventRecord(initialize_u_start1);
    initialize_u<<<numBlocks, threadsPerBlock>>>(d_u_cur, N, M);
    hipEventRecord(initialize_u_stop1);

    hipEventRecord(initialize_u_start2);
    initialize_u<<<numBlocks, threadsPerBlock>>>(d_u_prev, N, M);
    hipEventRecord(initialize_u_stop2);
    
    hipEventRecord(initialize_u_start3);
    initialize_u<<<numBlocks, threadsPerBlock>>>(d_u_next, N, M);
    hipEventRecord(initialize_u_stop3);
    
    hipEventRecord(define_c_start);
    define_c<<<numBlocks, threadsPerBlock>>>(d_c, c_max, N, M);
    hipEventRecord(define_c_stop);
    
    hipEventRecord(define_initial_condition_start); 
    define_initial_condition<<<numBlocks, threadsPerBlock>>>(d_u_cur, N, M, x0, y0);
    hipEventRecord(define_initial_condition_stop);

    float elapsedTimeFor, total = 0;

    for (int i = 0; i < N_time; i++)
    {
        hipEventRecord(calculate_wave_start);
        calculate_wave<<<numBlocks, threadsPerBlock>>>(d_u_next, d_u_cur, d_u_prev, d_c, N, M, dt, dh);
        hipEventRecord(calculate_wave_stop);
        hipEventSynchronize(calculate_wave_stop);
        hipEventElapsedTime(&elapsedTimeFor, calculate_wave_start, calculate_wave_stop);
        total +=elapsedTimeFor;

    }
    total /= N_time;

    print_time("fim :");
    hipEventSynchronize(define_initial_condition_stop);
    hipEventSynchronize(define_c_stop);
    hipEventSynchronize(initialize_u_stop3);
    hipEventSynchronize(initialize_u_stop2);
    hipEventSynchronize(initialize_u_stop1);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, define_initial_condition_start, define_initial_condition_stop);
    printf("define_initial_condition: %f ms\n", elapsedTime);
    hipEventElapsedTime(&elapsedTime, define_c_start, define_c_stop);
    printf("define_c: %f ms\n", elapsedTime);
    hipEventElapsedTime(&elapsedTime, initialize_u_start1, initialize_u_stop1); 
    printf("initialize_u1: %f ms\n", elapsedTime);
    hipEventElapsedTime(&elapsedTime, initialize_u_start2, initialize_u_stop2);
    printf("initialize_u2: %f ms\n", elapsedTime);
    hipEventElapsedTime(&elapsedTime, initialize_u_start3, initialize_u_stop3);
    printf("initialize_u3: %f ms\n", elapsedTime);
    printf("calculate_wave: %f ms\n", total);


    // Libera a memória
    hipFree(d_u_next);
    hipFree(d_u_cur);
    hipFree(d_u_prev);
    hipFree(d_c);
    free(time_arr);
    free(s_arr);
    free(h_u);


    return 0;
}
